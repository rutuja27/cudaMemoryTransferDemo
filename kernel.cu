#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include ""
#include <chrono>
#include <stdio.h>

#include <time.h>
#include <ctime> //defines localtime 
#include <chrono>

#include <fstream>

#ifdef WIN32
#include<windows.h>
#endif

//using namespace System;
using namespace std;

#if defined(_MSC_VER) || defined(_MSC_EXTENSIONS)
#define DELTA_EPOCH_IN_MICROSECS  11644473600000000Ui64
#else
#define DELTA_EPOCH_IN_MICROSECS  11644473600000000ULL
#endif

struct TimeStamp
{
    unsigned long long int seconds;
    unsigned int microSeconds;
};

const int rows = 2048;
const int cols = 1200;



class GetTime
{

public:

    unsigned long long secs;
    unsigned long long usec;
    time_t curr_time;
    //timeval tv;

    GetTime(long long unsigned int secs, long long unsigned int usec);

    struct timezone
    {
        int  tz_minuteswest; /* minutes W of Greenwich */
        int  tz_dsttime;     /* type of dst correction */
    };

    // Definition of a gettimeofday function
    //int getdaytime(struct timeval *tv, struct timezone *tz);
    std::chrono::system_clock::duration duration_since_midnight();
    TimeStamp getPCtime();

};

GetTime::GetTime(long long unsigned int sec, long long unsigned int usec)
{
    secs = secs;
    usec = usec;
}

std::chrono::system_clock::duration GetTime::duration_since_midnight()
{

    auto now = std::chrono::system_clock::now();
    time_t tnow = std::chrono::system_clock::to_time_t(now);
    tm *date = std::localtime(&tnow);
    date->tm_hour = 0;
    date->tm_min = 0;
    date->tm_sec = 0;
    auto midnight = std::chrono::system_clock::from_time_t(std::mktime(date));
    return now - midnight;

}


TimeStamp GetTime::getPCtime()
{

    auto since_midnight = duration_since_midnight();

    auto hours = std::chrono::duration_cast<std::chrono::hours>(since_midnight);
    auto minutes = std::chrono::duration_cast<std::chrono::minutes>(since_midnight - hours);
    auto seconds = std::chrono::duration_cast<std::chrono::seconds>(since_midnight - hours - minutes);
    auto milliseconds = std::chrono::duration_cast<std::chrono::milliseconds>(since_midnight - hours - minutes - seconds);
    auto microseconds = std::chrono::duration_cast<std::chrono::microseconds>(since_midnight - hours - minutes - seconds - milliseconds);
    auto nanoseconds = std::chrono::duration_cast<std::chrono::nanoseconds>(since_midnight - hours - minutes - seconds - milliseconds - microseconds);

    this->secs = (hours.count() * 3600 + minutes.count() * 60 + seconds.count());
    this->usec = (milliseconds.count() * 1000 + microseconds.count() + nanoseconds.count() / 1000);
    TimeStamp ts = { this->secs, uint64_t(this->usec) };

    return ts;
}

hipError_t addWithCuda(unsigned int rows, unsigned int cols, short *c, 
                        short *a, short *b);


__global__ void addKernel(short *c, short *a, short *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{

    short *a = new short[rows*cols*100];
    short *b = new short[rows*cols*100];
    short *c = new short[rows*cols*100];
    unsigned int size = rows * cols * 100;

    for (int i = 0; i < size; i++) {

        a[i] = 0;
        b[i] = 0;
        c[i] = 0;
    }
    
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(rows, cols, c, a, b);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        getchar();
        return 1;
    }
    else {
        fprintf(stderr, "addWithCuda Succeded");
    }


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    getchar();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(unsigned int rows, unsigned int cols, short *c, short *a, short *b)
{
    short *dev_a = 0;
    short *dev_b = 0;
    short *dev_c = 0;
    hipError_t cudaStatus;
    unsigned int size = rows * cols * 100;
    
    GetTime mem_time(0, 0);
    TimeStamp start, end;
    float elapsed;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(short));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(short));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;

    }
    start = mem_time.getPCtime();
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(short));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    end = mem_time.getPCtime();
    elapsed = ((end.seconds * 1e6 + end.microSeconds) - (start.seconds * 1e6 + start.microSeconds)) / 1e6;
    printf(" Time taken for mem alloc %lf", elapsed);

    start = mem_time.getPCtime();
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(short), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    end = mem_time.getPCtime();
    elapsed = (((end.seconds * 1e6) + end.microSeconds) - ((start.seconds * 1e6) + start.microSeconds)) / 1e6;
    printf(" Time taken for mem transfer host to device %lf\n", elapsed);

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(short), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    cudaStatus = hipMemcpy(dev_c, c, size * sizeof(short), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    // Launch a kernel on the GPU with one thread for each element.
    dim3 grid(640, 750);
    dim3 block(32, 16);
    addKernel<<<grid, block>>>(dev_c, dev_a, dev_b);
   
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    
    start = mem_time.getPCtime();
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(short), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    end = mem_time.getPCtime();
    elapsed = (((end.seconds * 1e6) + end.microSeconds) - ((start.seconds * 1e6) + start.microSeconds)) / 1e6;
    printf(" Time taken for mem transfer from dev to host %lf\n", elapsed);
    

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
